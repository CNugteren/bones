#include "hip/hip_runtime.h"
/* STARTDEF
void bones_prekernel_<algorithm_name>_0(<devicedefinitions>, <argument_definition>);
ENDDEF */
// Start of the <algorithm_name> kernel
__global__ void bones_kernel_<algorithm_name>_0(<devicedefinitions>, <argument_definition>) {
  const int bones_global_id = blockIdx.x*blockDim.x + threadIdx.x;
  if (bones_global_id < (<parallelism>)) {
    
    // Calculate the global ID(s) based on the thread id
    <ids>
    
    // Start the computation
<algorithm_code1>
  }
}

// Function to start the kernel
extern "C" void bones_prekernel_<algorithm_name>_0(<devicedefinitions>, <argument_definition>) {
  int bones_block_size;
  if      (<parallelism> >= 64*512) { bones_block_size = 512;}
  else if (<parallelism> >= 64*256) { bones_block_size = 256;}
  else if (<parallelism> >= 64*128) { bones_block_size = 128;}
  else if (<parallelism> >= 64*64 ) { bones_block_size = 64; }
  else { bones_block_size = 32; }
  dim3 bones_threads(bones_block_size);
  dim3 bones_grid(DIV_CEIL(<parallelism>,bones_block_size));
  bones_kernel_<algorithm_name>_0<<< bones_grid, bones_threads >>>(<names>, <argument_name>);
}
